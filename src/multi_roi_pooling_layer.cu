#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Written by liyong (yong.li@vipl.ict.ac.cn), gpu version for PG-CNN
// Given a [N C H W] input, we aim to get top_num branch, each with the shape of  [N  C  pool_h  pool_w].
// Step:
//      1, crop a sub-feature map with the shape of [1  c  pool_h  pool_w] according to the facial landmarks for each image
//      2, for each branch, concatenate the N sub-feature map from the N images,
//         obtaining the resulted feature map, with the shape of [N c pool_h pool_w]

// Details: when cropping the sub-feature map, we set 0 to the related coordinates of the croped feature map, 
//          if the cropping index was checked to be smaller than 0 or larger than the spatial dimension of the input feature map.
// ------------------------------------------------------------------

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/multi_roi_pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;
using std::min;
using std::floor;
using std::ceil;

namespace caffe {

template <typename Dtype>
__global__ void MultiROIPoolForward(const int nthreads, const Dtype* bottom_data,
        const int batch_size, const int channels, const int height,
        const int width, const int pooled_height, const int pooled_width,
        const Dtype* bottom_rois, int num_rois, Dtype* top_data, const int top_idx, const Dtype scale) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        // we have lanuch roi_num * batch_size * channel * pooled_height thread, so can get
        int c = index % channels;
        int batch_idx = (index / channels) % batch_size;

        // infer coordinate idx based on top_idx
        int roi_idx = (batch_idx * num_rois + top_idx) * 2;

        // 2 exception: 1 negative start point , 2 exceed boundary of bottom
        // for netative start point, we will copy 0-index from bottom to and abs(corrdinate) in top
        // for exceed condition, we will copy roi_coordinate from bottom to and 0-index in top

        int roi_start_w = round(bottom_rois[roi_idx + 0] * scale - pooled_width/2); // corrdinate begin with 0
        int roi_start_h = round(bottom_rois[roi_idx + 1] * scale - pooled_height/2);

        int bottom_offset_w = roi_start_w >= 0 ? roi_start_w : 0;
        int bottom_offset_h = roi_start_h >= 0 ? roi_start_h : 0;

        int top_offset_w = roi_start_w >= 0 ? 0: abs(roi_start_w);
        int top_offset_h = roi_start_h >= 0 ? 0: abs(roi_start_h);

        int copy_len_w = 0;
        if(roi_start_w < 0) {
            copy_len_w = pooled_width - abs(roi_start_w);
        } else if(pooled_width + bottom_offset_w > width) {
            copy_len_w = width - bottom_offset_w;
        } else {
            copy_len_w = pooled_width;
        }

        int copy_len_h = 0;
        if(roi_start_h < 0) {
            copy_len_h = pooled_height - abs(roi_start_h);
        } else if(pooled_height + bottom_offset_h > height) {
            copy_len_h = height - bottom_offset_h;
        } else {
            copy_len_h = pooled_height;
        }


        for (int ph = 0; ph < copy_len_h; ++ph) {
            // copy it per width
            int bottom_data_idx = ((batch_idx * channels + c) * 
                    height + bottom_offset_h + ph) * width + bottom_offset_w;
            int top_data_idx = ((batch_idx * channels + c) * pooled_height + top_offset_h + ph) * 
                pooled_width + top_offset_w;

            for (int pw = 0; pw < copy_len_w; ++pw) {
                top_data[top_data_idx + pw] = bottom_data[bottom_data_idx + pw];
            }
        }
    }
}

template <typename Dtype>
__global__ void MultiROIPoolBackward(const int nthreads, const Dtype* top_diff,
    const int top_idx, const int num_rois, const int batch_size,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, Dtype* bottom_diff, const Dtype* bottom_rois, const Dtype scale) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        // we have lanuch roi_num * batch_size * channel * pooled_height thread, so can get
        int c = index % channels;
        int batch_idx = (index / channels) % batch_size;
        
        // infer coordinate idx based on top_idx
        int roi_idx = (batch_idx * num_rois + top_idx) * 2;

        int roi_start_w = round(bottom_rois[roi_idx + 0] * scale - pooled_width/2); // corrdinate begin with 0
        int roi_start_h = round(bottom_rois[roi_idx + 1] * scale - pooled_height/2);

        int bottom_offset_w = roi_start_w >= 0 ? roi_start_w : 0;
        int bottom_offset_h = roi_start_h >= 0 ? roi_start_h : 0;

        int top_offset_w = roi_start_w >= 0 ? 0: abs(roi_start_w);
        int top_offset_h = roi_start_h >= 0 ? 0: abs(roi_start_h);

        int copy_len_w = 0;
        if(roi_start_w < 0) {
            copy_len_w = pooled_width - abs(roi_start_w);
        } else if(pooled_width + bottom_offset_w > width) {
            copy_len_w = width - bottom_offset_w;
        } else {
            copy_len_w = pooled_width;
        }

        int copy_len_h = 0;
        if(roi_start_h < 0) {
            copy_len_h = pooled_height - abs(roi_start_h);
        } else if(pooled_height + bottom_offset_h > height) {
            copy_len_h = height - bottom_offset_h;
        } else {
            copy_len_h = pooled_height;
        }

        for (int ph = 0; ph < copy_len_h; ++ph) {
            // copy it per width
            int bottom_diff_idx = ((batch_idx * channels + c) * 
                    height + bottom_offset_h + ph) * width + bottom_offset_w;
            int top_diff_idx = ((batch_idx * channels + c) * pooled_height + top_offset_h) * pooled_width + top_offset_w;

            for (int pw = 0; pw < copy_len_w; ++pw) {
                bottom_diff[bottom_diff_idx + pw] += top_diff[top_diff_idx + pw];
            }
        }
    }
}

template <typename Dtype>
void MultiROIPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  // roi param:
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  // Number of ROIs
  int num_rois = bottom[1]->count(1,2); // about 20
  int batch_size = bottom[0]->num();
  int top_count = top[0]->count();
  for(int idx = 0; idx < top_num; idx ++) {
      Dtype* top_data = top[idx]->mutable_gpu_data();
      caffe_gpu_set(top_count, Dtype(0), top_data);
  }
  // 
  const int count = batch_size * channels_;
  // note we have to crop from each [C H W] and get a roi[1 c 6 6], concat N roi to get a top
  // try to lanuch batch_size * channel threads
  for(int top_idx = 0; top_idx < num_rois; top_idx ++){
      Dtype *top_data = top[top_idx]->mutable_gpu_data();
      MultiROIPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
              count, bottom_data, batch_size, channels_, height_, width_,
              pooled_height_, pooled_width_, bottom_rois, top_num, top_data, top_idx, spatial_scale_);
  }
}

template <typename Dtype>
void MultiROIPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
        const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    const Dtype* bottom_rois = bottom[1]->gpu_data();
    Dtype *bottom_diff = bottom[0]->mutable_gpu_diff();
    int num_rois = bottom[1]->count(1,2);
    int bottom_count = bottom[0]->count();
    int batch_size = bottom[0]->num();

    caffe_gpu_set(bottom_count, Dtype(0), bottom_diff);

    const int count = batch_size * channels_;
    for(int top_idx = 0; top_idx < num_rois; top_idx ++){
        const Dtype *top_diff = top[top_idx]->gpu_diff();
        MultiROIPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
                count, top_diff,
                top_idx, num_rois, batch_size,
                channels_, height_, width_,
                pooled_height_, pooled_width_, bottom_diff, bottom_rois, spatial_scale_);
    }
}


INSTANTIATE_LAYER_GPU_FUNCS(MultiROIPoolingLayer);

}  // namespace caffe
